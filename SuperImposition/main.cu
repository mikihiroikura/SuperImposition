#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#define _USE_MATH_DEFINES

#include <algorithm>            // std::min, std::max
#include <opencv2/opencv.hpp>
#include <iostream>
#include <Windows.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtc/matrix_transform.hpp> 
#include <glm/gtx/transform.hpp>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

// imgui include
#include "imgui/imgui.h"
#include "imgui/imgui_impl_opengl3.h"
#include "imgui/imgui_impl_glfw.h"

#include <librealsense2/rs.hpp>
#include "realsense_glfw.h"


#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

using namespace std;

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1280;
const unsigned int window_height = 720;

const unsigned int mesh_width = 106*8;
const unsigned int mesh_height = 60*8;

GLuint Matrix;

//shader object
GLFWwindow* window;
static GLuint vertShader;
static GLuint fragShader;
GLuint gl2Program;

GLuint vao;

// vbo variables
GLuint vbo, tcbo;
struct hipGraphicsResource* cuda_vbo_resource, *cuda_tcbo_resource;

float g_fAnim = 0.0;

#pragma warning(disable:4996)

// mouse controls
float rotate_x = 0.0, rotate_y = 0.0;
float translate_x = 0.0, translate_y = 0.0, translate_z = -.0;
double mouse_x, mouse_y, mouse_x_old, mouse_y_old;
double horiz_angle = -M_PI, vert_angle = 0.0;
double mouse_speed = 0.01;
double dx = 0.0, dy = 0.0;
float init_fov = 60;
float fov = init_fov;
glm::vec3 position(0, 0, -1);
glm::vec3 up(0, -1, 0);
glm::vec3 direction(0, 0, 0);
bool hovered;
float Time = 0;
float pointsize = 2.5;
glm::mat4 mvp, Model, View, Projection;
GLint matlocation;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int* pArgc = NULL;
char** pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

// Realsense
rs2::pointcloud pc;
rs2::points points;
rs2::pipeline pipe;
rs2::frameset frames;
texture_gl tex;
const rs2::vertex* h_vertices;
float3* h_vertices_f;
const rs2::texture_coordinate* tex_coords;
// map OpenGL buffer object for writing from CUDA
//const rs2::vertex* d_vertices;
float3* d_vertices;
const rs2::vertex* gl_vertices;
unsigned int size_vert;

////////////////////////////////////////////////////////////////////////////////
// GL functionality
bool initGL(int* argc, char** argv);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);
static void setfov(GLFWwindow* window, double x, double y);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource);

const char* sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float3* vertex, unsigned int width, unsigned int height, float rot_x, float rot_y, float trans_z, float times)
{
    //��������ʒu
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    //���W�ϊ�


    // write output vertex
    vertex[y * width + x] = make_float3(vertex[y * width + x].x, vertex[y * width + x].y, vertex[y * width + x].z);
}


void launch_kernel(float3* vertex, float rot_x, float rot_y, float trans_z, float times)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel << < grid, block >> > (vertex, mesh_width, mesh_height, rot_x, rot_y, trans_z, times);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    char* ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char**)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char**)argv, "file", (char**)&ref_file);
        }
    }

    printf("\n");

    pipe.start();
    for (size_t i = 0; i < 30; i++)
    {
        frames = pipe.wait_for_frames();
    }

    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (false == initGL(&argc, argv))
    {
        return false;
    }

    //// run the cuda part
    //runCuda(&cuda_vbo_resource);

    //�����Ƀ��[�v������
    while (glfwWindowShouldClose(window) == GL_FALSE)
    {
        // run the cuda part
        /*runCuda(&cuda_vbo_resource);*/

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        gluPerspective(fov, (GLfloat)window_width / (GLfloat)window_height, 0.1f, 100.0f);

        glMatrixMode(GL_MODELVIEW);
        glPushMatrix();
        position = glm::vec3(cos(vert_angle) * sin(horiz_angle), sin(vert_angle), cos(vert_angle) * cos(horiz_angle));
        gluLookAt(position.x, position.y, position.z, 0, 0, 1, 0, -1, 0); //����ŃJ�����̏�����̎���--y�����ɂ��邱�Ƃŏ㉺�����킹��
        glRotated(rotate_x, 1, 0, 0);
        glRotated(rotate_y, 0, 1, 0);
        glTranslatef(translate_x, 0, 0);
        glTranslatef(0, translate_y, 0);
        glTranslatef(0, 0, translate_z);

        //�J�[�\���ʒu����ړ��ω��ʂ��v�Z
        glfwGetCursorPos(window, &mouse_x, &mouse_y);
        dx = mouse_x - mouse_x_old;
        dy = mouse_y - mouse_y_old;

        //���N���b�N���Ă���΂���IMGUI���Window�ɂ��Ȃ���΁C�ړ��ω��ʂ���Ɋp�x�X�V
        if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT) == GLFW_PRESS && !hovered)
        {
            horiz_angle += mouse_speed * dx;
            vert_angle += mouse_speed * dy;
        }
        mouse_x_old = mouse_x;
        mouse_y_old = mouse_y;

        //�X�y�[�X�L�[�������Ă���΁C�p�����[�^���Z�b�g
        if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        {
            horiz_angle = -M_PI;
            vert_angle = 0.0;
            rotate_x = 0.0, rotate_y = 0.0;
            translate_x = 0.0, translate_y = 0.0, translate_z = -.0;
            fov = init_fov;
        }

        //realsense
        auto frames = pipe.wait_for_frames();
        auto color = frames.get_color_frame();
        pc.map_to(color);
        auto depth = frames.get_depth_frame();
        points = pc.calculate(depth);
        tex.upload(color);
        h_vertices = points.get_vertices();
        h_vertices_f = (float3*)h_vertices;
        hipMemcpy(d_vertices, h_vertices_f, size_vert, hipMemcpyHostToDevice);//�f�o�C�X�������Ƀz�X�g�������̒l���R�s�[

        runCuda(&cuda_vbo_resource);
        draw_pointcloud2(&vbo, tex_coords, &tcbo, window_width, window_height, tex, points, translate_z, rotate_x, rotate_y);

        glPopMatrix();

        glfwPollEvents();

        //start imgui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::SetNextWindowSize(ImVec2(320, 300), ImGuiCond_Once);
        ImGui::Begin("hello world");
        ImGui::Text("This is useful text");
        hovered = ImGui::IsWindowHovered(ImGuiHoveredFlags_AllowWhenBlockedByActiveItem); //IMGUI���Window�ł̃J�[�\���������̃t���O�𗧂Ă�
        ImGui::DragFloat("rotate x", &rotate_x);
        ImGui::DragFloat("rotate y", &rotate_y);
        ImGui::DragFloat("trans x", &translate_x);
        ImGui::DragFloat("trans y", &translate_y);
        ImGui::DragFloat("trans z", &translate_z);
        ImGui::End();

        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());


        glfwSwapBuffers(window);

        g_fAnim += 0.01f;
    }

    deleteVBO(&vbo, cuda_vbo_resource);
    glDeleteBuffers(1, &tcbo);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    //GLFW�̏�����
    if (glfwInit() == GL_FALSE)
    {
        std::cerr << "Can't initilize GLFW" << std::endl;
        return 1;
    }

    //Window�̍쐬
    window = glfwCreateWindow(window_width, window_height, "Cuda GL Interop (VBO)", NULL, NULL);
    if (window == nullptr)
    {
        std::cerr << "Can't create GLFW window." << std::endl;
        glfwTerminate();
        return 1;
    }
    
    //Window��OpenGL�̑Ώۂɂ���
    glfwMakeContextCurrent(window);
    //MakeCOntextcurrent�̌�ɍs��Ȃ��Ǝ��s����炵��
    if (glewInit() != GLEW_OK)
    {
        std::cerr << "Can't initilize GLEW" << std::endl;
        return 1;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);   //�w�i�F�̎w��
    glDisable(GL_DEPTH_TEST);
   
    //���_�o�b�t�@�I�u�W�F�N�g
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    size_vert = 407040 * 3 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size_vert, h_vertices_f, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, hipGraphicsRegisterFlagsNone));//CUDA�̃O���t�B�b�N�X���\�[�X�ɓo�^����

    //Tex coordinate�o�b�t�@�I�u�W�F�N�g
    glGenBuffers(1, &tcbo);
    glBindBuffer(GL_ARRAY_BUFFER, tcbo);
    unsigned int size_uv = 407040 * 2 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size_uv, tex_coords, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    //�X�N���[������Callback����֐��̎w��
    glfwSetScrollCallback(window, setfov);

    //imgui�̏�����
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource)
{
    
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_vertices, &num_bytes,
        *vbo_resource));

    launch_kernel(d_vertices, rotate_x, rotate_y, translate_z, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

static void setfov(GLFWwindow* window, double x, double y) {
    fov -= static_cast<GLfloat>(y);
}
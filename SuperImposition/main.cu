#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <algorithm>            // std::min, std::max
#include <opencv2/opencv.hpp>
#include <iostream>
#include <Windows.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtc/matrix_transform.hpp> 
#include <glm/gtx/transform.hpp>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

// imgui include
#include "imgui/imgui.h"
#include "imgui/imgui_impl_opengl3.h"
#include "imgui/imgui_impl_glfw.h"

#include <librealsense2/rs.hpp>
#include "realsense_glfw.hpp"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

using namespace std;

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1280;
const unsigned int window_height = 720;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

glm::mat4 mvp;
GLuint Matrix;

//shader object
GLFWwindow* window;
static GLuint vertShader;
static GLuint fragShader;
GLuint gl2Program;

GLuint vao;

// vbo variables
GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;
void* d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int* pArgc = NULL;
char** pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

// Realsense
rs2::pointcloud pc;
rs2::points points;
rs2::pipeline pipe;
rs2::frameset frames;
texture_gl tex;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char** argv, char* ref_file);

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res,
    unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// rendering callbacks
int readShaderSource(GLuint shader, const char* file);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource);

const char* sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    //���W�ϊ�
    float ud = cosf(time) * u + sinf(time) * v;
    float vd = -sinf(time) * u + cosf(time) * v;


    // write output vertex
    pos[y * width + x] = make_float4(ud, w, vd, 1.0f);
}


void launch_kernel(float4* pos, unsigned int mesh_width,
    unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    char* ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char**)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char**)argv, "file", (char**)&ref_file);
        }
    }

    printf("\n");

    pipe.start();
    for (size_t i = 0; i < 30; i++)
    {
        frames = pipe.wait_for_frames();
    }

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    //GLFW�̏�����
    if (glfwInit() == GL_FALSE)
    {
        std::cerr << "Can't initilize GLFW" << std::endl;
        return 1;
    }

    //Window�̍쐬
    window = glfwCreateWindow(window_width, window_height, "Cuda GL Interop (VBO)", NULL, NULL);
    if (window == nullptr)
    {
        std::cerr << "Can't create GLFW window." << std::endl;
        glfwTerminate();
        return 1;
    }
    
    //Window��OpenGL�̑Ώۂɂ���
    glfwMakeContextCurrent(window);
    //MakeCOntextcurrent�̌�ɍs��Ȃ��Ǝ��s����炵��
    if (glewInit() != GLEW_OK)
    {
        std::cerr << "Can't initilize GLEW" << std::endl;
        return 1;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);   //�w�i�F�̎w��
    glDisable(GL_DEPTH_TEST);
    
    //vertShader = glCreateShader(GL_VERTEX_SHADER);
    //fragShader = glCreateShader(GL_FRAGMENT_SHADER);

    //////�\�[�X�v���O�����ǂݍ���
    //if (readShaderSource(vertShader, "points.vert")) exit(1);
    //if (readShaderSource(fragShader, "points.frag")) exit(1);

    //////Shader�R���p�C��
    //glCompileShader(vertShader);
    //glCompileShader(fragShader);

    //////�v���O�����I�u�W�F�N�g�̍쐬
    //gl2Program = glCreateProgram();
    //glAttachShader(gl2Program, vertShader);
    //glAttachShader(gl2Program, fragShader);
    //glDeleteShader(vertShader);
    //glDeleteShader(fragShader);

    ////�v���O�����I�u�W�F�N�g�̃����N
    //glBindAttribLocation(gl2Program, 0, "position");
    //glBindFragDataLocation(gl2Program, 0, "gl_FragColor");
    //glLinkProgram(gl2Program);

    ////���_�z��I�u�W�F�N�g
    //glGenVertexArrays(1, &vao);
    //glBindVertexArray(vao);

    //���_�o�b�t�@�I�u�W�F�N�g
    /*glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, nullptr, GL_DYNAMIC_DRAW);*/

    ////Vertexshader�̎Q��
    //glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
    //glEnableVertexAttribArray(0);

    ////���_�o�b�t�@�I�u�W�F�N�g�̌�������
    //glBindBuffer(GL_ARRAY_BUFFER, 0);
    //glBindVertexArray(0);


    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    // �J�����s��
    //glm::mat4 View = glm::lookAt(
    //    glm::vec3(4, 4, 4), // ���[���h��ԂŃJ������(4,3,3)�ɂ���܂��B
    //    glm::vec3(0, 0, 0), // ���_�����Ă��܂��B
    //    glm::vec3(0, 1, 0)  // ���������(0,-1,0�ɃZ�b�g����Ə㉺�t�]���܂��B)
    //);
    //glm::mat4 Projection = glm::perspective(glm::radians(60.0f), 4.0f / 3.0f, 0.1f, 10.0f);
    //mvp = Projection;// *View;

    //Matrix = glGetUniformLocation(gl2Program, "MVP");

    //imgui�̏�����
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init();

    return true;
}

/*
** �V�F�[�_�[�̃\�[�X�v���O�������������ɓǂݍ���
*/
int readShaderSource(GLuint shader, const char* file)
{
    FILE* fp;
    const char* source;
    GLsizei length;
    int ret;

    /* �t�@�C�����J�� */
    fp = fopen(file, "rb");
    if (fp == NULL) {
        perror(file);
        return -1;
    }

    /* �t�@�C���̖����Ɉړ������݈ʒu (�܂�t�@�C���T�C�Y) �𓾂� */
    fseek(fp, 0L, SEEK_END);
    length = ftell(fp);

    /* �t�@�C���T�C�Y�̃��������m�� */
    source = (char*)malloc(length);
    if (source == NULL) {
        fprintf(stderr, "Could not allocate read buffer.\n");
        return -1;
    }

    /* �t�@�C����擪����ǂݍ��� */
    fseek(fp, 0L, SEEK_SET);
    ret = fread((void*)source, 1, length, fp) != (size_t)length;
    fclose(fp);

    /* �V�F�[�_�̃\�[�X�v���O�����̃V�F�[�_�I�u�W�F�N�g�ւ̓ǂݍ��� */
    if (ret)
        fprintf(stderr, "Could not read file: %s.\n", file);
    else
        glShaderSource(shader, 1, &source, &length);

    /* �m�ۂ����������̊J�� */
    free((void*)source);

    return ret;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char** argv, char* ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (false == initGL(&argc, argv))
    {
        return false;
    }

    //// create VBO
    //createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    //// run the cuda part
    //runCuda(&cuda_vbo_resource);

    //�����Ƀ��[�v������
    while (glfwWindowShouldClose(window) == GL_FALSE)
    {
        // run the cuda part
        /*runCuda(&cuda_vbo_resource);*/

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        glLoadIdentity();
        glMatrixMode(GL_MODELVIEW);
        glPushMatrix();
        gluLookAt(0, 0, 0, 0, 0, 1, 0, -1, 0); //����ŃJ�����̏�����̎���--y�����ɂ��邱�Ƃŏ㉺�����킹��
        glRotated(rotate_x, 1, 0, 0);
        glRotated(rotate_y, 0, 1, 0);
        glTranslatef(0, 0, translate_z);

        //realsense
        auto frames = pipe.wait_for_frames();
        auto color = frames.get_color_frame();
        pc.map_to(color);
        auto depth = frames.get_depth_frame();
        points = pc.calculate(depth);
        tex.upload(color);
        draw_pointcloud(window_width, window_height, tex, points, translate_z, rotate_x, rotate_y);

        /*glBindBuffer(GL_ARRAY_BUFFER, vbo);
        glVertexPointer(4, GL_FLOAT, 0, 0);
        glEnableClientState(GL_VERTEX_ARRAY);
        glColor3f(1.0, 0.0, 0.0);
        glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
        glDisableClientState(GL_VERTEX_ARRAY);*/
        glPopMatrix();

        glfwPollEvents();

        //start imgui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::Begin("hello world");
        ImGui::Text("This is useful text");
        ImGui::DragFloat("rotate x", &rotate_x);
        ImGui::DragFloat("rotate y", &rotate_y);
        ImGui::DragFloat("trans z", &translate_z);
        ImGui::End();

        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());


        glfwSwapBuffers(window);

        g_fAnim += 0.01f;
    }

    deleteVBO(&vbo, cuda_vbo_resource);

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4* dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes,
        *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res,
    unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}


//////////////////////////////////////////////////////////////////////////////////
////! Display callback
//////////////////////////////////////////////////////////////////////////////////
//void display()
//{
//    sdkStartTimer(&timer);
//
//    // run CUDA kernel to generate vertex positions
//    runCuda(&cuda_vbo_resource);
//
//    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//
//    // set view matrix
//    glMatrixMode(GL_MODELVIEW);
//    glLoadIdentity();
//    glTranslatef(0.0, 0.0, translate_z);
//    glRotatef(rotate_x, 1.0, 0.0, 0.0);
//    glRotatef(rotate_y, 0.0, 1.0, 0.0);
//
//    // render from the vbo
//    glBindBuffer(GL_ARRAY_BUFFER, vbo);
//    glVertexPointer(4, GL_FLOAT, 0, 0);
//
//    glEnableClientState(GL_VERTEX_ARRAY);
//    glColor3f(1.0, 0.0, 0.0);
//    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
//    glDisableClientState(GL_VERTEX_ARRAY);
//
//    glutSwapBuffers();
//
//    g_fAnim += 0.01f;
//
//    sdkStopTimer(&timer);
//    computeFPS();
//}
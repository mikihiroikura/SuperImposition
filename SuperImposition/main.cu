#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <algorithm>            // std::min, std::max
#include <opencv2/opencv.hpp>
#include <iostream>
#include <Windows.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtc/matrix_transform.hpp> 
#include <glm/gtx/transform.hpp>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <hip/hip_vector_types.h>

// imgui include
#include "imgui/imgui.h"
#include "imgui/imgui_impl_opengl3.h"
#include "imgui/imgui_impl_glfw.h"

#include <librealsense2/rs.hpp>
#include "realsense_glfw.h"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

using namespace std;

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1280;
const unsigned int window_height = 720;

const unsigned int mesh_width = 106*8;
const unsigned int mesh_height = 60*8;

glm::mat4 mvp;
GLuint Matrix;

//shader object
GLFWwindow* window;
static GLuint vertShader;
static GLuint fragShader;
GLuint gl2Program;

GLuint vao;

// vbo variables
GLuint vbo, tcbo;
struct hipGraphicsResource* cuda_vbo_resource, *cuda_tcbo_resource;
void* d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int* pArgc = NULL;
char** pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

// Realsense
rs2::pointcloud pc;
rs2::points points;
rs2::pipeline pipe;
rs2::frameset frames;
texture_gl tex;
const rs2::vertex* vertices;
const rs2::texture_coordinate* tex_coords;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char** argv, char* ref_file);

// GL functionality
bool initGL(int* argc, char** argv);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource, const rs2::vertex* vertex);

const char* sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float3* pos, const rs2::vertex* vertex, unsigned int width, unsigned int height, float rot_x, float rot_y, float trans_z)
{
    //��������ʒu
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    //���W�ϊ�
    


    // write output vertex
    //pos[y * width + x] = make_float3((vertex+ y * width + x)->x, (vertex + y * width + x)->y, (vertex + y * width + x)->z);
    pos[y * width + x] = make_float3(x*0.01, y*0.01, 0);
}


void launch_kernel(float3* pos, const rs2::vertex* vertex, float rot_x, float rot_y, float trans_z)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel << < grid, block >> > (pos, vertex, mesh_width, mesh_height, rot_x, rot_y, trans_z);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    char* ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char**)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char**)argv, "file", (char**)&ref_file);
        }
    }

    printf("\n");

    pipe.start();
    for (size_t i = 0; i < 30; i++)
    {
        frames = pipe.wait_for_frames();
    }

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    //GLFW�̏�����
    if (glfwInit() == GL_FALSE)
    {
        std::cerr << "Can't initilize GLFW" << std::endl;
        return 1;
    }

    //Window�̍쐬
    window = glfwCreateWindow(window_width, window_height, "Cuda GL Interop (VBO)", NULL, NULL);
    if (window == nullptr)
    {
        std::cerr << "Can't create GLFW window." << std::endl;
        glfwTerminate();
        return 1;
    }
    
    //Window��OpenGL�̑Ώۂɂ���
    glfwMakeContextCurrent(window);
    //MakeCOntextcurrent�̌�ɍs��Ȃ��Ǝ��s����炵��
    if (glewInit() != GLEW_OK)
    {
        std::cerr << "Can't initilize GLEW" << std::endl;
        return 1;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);   //�w�i�F�̎w��
    glDisable(GL_DEPTH_TEST);
    
    //vertShader = glCreateShader(GL_VERTEX_SHADER);
    //fragShader = glCreateShader(GL_FRAGMENT_SHADER);

    //////�\�[�X�v���O�����ǂݍ���
    //if (readShaderSource(vertShader, "points.vert")) exit(1);
    //if (readShaderSource(fragShader, "points.frag")) exit(1);

    //////Shader�R���p�C��
    //glCompileShader(vertShader);
    //glCompileShader(fragShader);

    //////�v���O�����I�u�W�F�N�g�̍쐬
    //gl2Program = glCreateProgram();
    //glAttachShader(gl2Program, vertShader);
    //glAttachShader(gl2Program, fragShader);
    //glDeleteShader(vertShader);
    //glDeleteShader(fragShader);

    ////�v���O�����I�u�W�F�N�g�̃����N
    //glBindAttribLocation(gl2Program, 0, "position");
    //glBindFragDataLocation(gl2Program, 0, "gl_FragColor");
    //glLinkProgram(gl2Program);

    ////���_�z��I�u�W�F�N�g
    //glGenVertexArrays(1, &vao);
    //glBindVertexArray(vao);

    //���_�o�b�t�@�I�u�W�F�N�g
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    unsigned int size_vert = 407040 * 3 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size_vert, vertices, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, hipGraphicsRegisterFlagsWriteDiscard));//CUDA�̃O���t�B�b�N�X���\�[�X�ɓo�^����

    //Tex coordinate�o�b�t�@�I�u�W�F�N�g
    glGenBuffers(1, &tcbo);
    glBindBuffer(GL_ARRAY_BUFFER, tcbo);
    unsigned int size_uv = 407040 * 2 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size_uv, tex_coords, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_tcbo_resource, tcbo, hipGraphicsRegisterFlagsWriteDiscard));//CUDA�̃O���t�B�b�N�X���\�[�X�ɓo�^����

    ////Vertexshader�̎Q��
    //glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
    //glEnableVertexAttribArray(0);

    ////���_�o�b�t�@�I�u�W�F�N�g�̌�������
    //glBindBuffer(GL_ARRAY_BUFFER, 0);
    //glBindVertexArray(0);

    
    


    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    // �J�����s��
    //glm::mat4 View = glm::lookAt(
    //    glm::vec3(4, 4, 4), // ���[���h��ԂŃJ������(4,3,3)�ɂ���܂��B
    //    glm::vec3(0, 0, 0), // ���_�����Ă��܂��B
    //    glm::vec3(0, 1, 0)  // ���������(0,-1,0�ɃZ�b�g����Ə㉺�t�]���܂��B)
    //);
    //glm::mat4 Projection = glm::perspective(glm::radians(60.0f), 4.0f / 3.0f, 0.1f, 10.0f);
    //mvp = Projection;// *View;

    //Matrix = glGetUniformLocation(gl2Program, "MVP");

    //imgui�̏�����
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char** argv, char* ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (false == initGL(&argc, argv))
    {
        return false;
    }

    //// run the cuda part
    //runCuda(&cuda_vbo_resource);

    //�����Ƀ��[�v������
    while (glfwWindowShouldClose(window) == GL_FALSE)
    {
        // run the cuda part
        /*runCuda(&cuda_vbo_resource);*/

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        glLoadIdentity();
        glMatrixMode(GL_MODELVIEW);
        glPushMatrix();
        gluLookAt(0, 0, 0, 0, 0, 1, 0, -1, 0); //����ŃJ�����̏�����̎���--y�����ɂ��邱�Ƃŏ㉺�����킹��
        glRotated(rotate_x, 1, 0, 0);
        glRotated(rotate_y, 0, 1, 0);
        glTranslatef(0, 0, translate_z);

        //realsense
        auto frames = pipe.wait_for_frames();
        auto color = frames.get_color_frame();
        pc.map_to(color);
        auto depth = frames.get_depth_frame();
        points = pc.calculate(depth);
        tex.upload(color);
        vertices = points.get_vertices();
        runCuda(&cuda_vbo_resource, vertices);
        //draw_pointcloud(vertices, &vbo, tex_coords, &tcbo, window_width, window_height, tex, points, translate_z, rotate_x, rotate_y);
        draw_pointcloud2(&vbo, tex_coords, &tcbo, window_width, window_height, tex, points, translate_z, rotate_x, rotate_y);

        /*glBindBuffer(GL_ARRAY_BUFFER, vbo);
        glVertexPointer(4, GL_FLOAT, 0, 0);
        glEnableClientState(GL_VERTEX_ARRAY);
        glColor3f(1.0, 0.0, 0.0);
        glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
        glDisableClientState(GL_VERTEX_ARRAY);*/
        glPopMatrix();

        glfwPollEvents();

        //start imgui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::Begin("hello world");
        ImGui::Text("This is useful text");
        ImGui::DragFloat("rotate x", &rotate_x);
        ImGui::DragFloat("rotate y", &rotate_y);
        ImGui::DragFloat("trans z", &translate_z);
        ImGui::End();

        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());


        glfwSwapBuffers(window);

        g_fAnim += 0.01f;
    }

    deleteVBO(&vbo, cuda_vbo_resource);

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource, const rs2::vertex* vertex)
{
    // map OpenGL buffer object for writing from CUDA
    float3* dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes,
        *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, vertex, rotate_x, rotate_y, translate_z);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}


//////////////////////////////////////////////////////////////////////////////////
////! Display callback
//////////////////////////////////////////////////////////////////////////////////
//void display()
//{
//    sdkStartTimer(&timer);
//
//    // run CUDA kernel to generate vertex positions
//    runCuda(&cuda_vbo_resource);
//
//    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//
//    // set view matrix
//    glMatrixMode(GL_MODELVIEW);
//    glLoadIdentity();
//    glTranslatef(0.0, 0.0, translate_z);
//    glRotatef(rotate_x, 1.0, 0.0, 0.0);
//    glRotatef(rotate_y, 0.0, 1.0, 0.0);
//
//    // render from the vbo
//    glBindBuffer(GL_ARRAY_BUFFER, vbo);
//    glVertexPointer(4, GL_FLOAT, 0, 0);
//
//    glEnableClientState(GL_VERTEX_ARRAY);
//    glColor3f(1.0, 0.0, 0.0);
//    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
//    glDisableClientState(GL_VERTEX_ARRAY);
//
//    glutSwapBuffers();
//
//    g_fAnim += 0.01f;
//
//    sdkStopTimer(&timer);
//    computeFPS();
//}